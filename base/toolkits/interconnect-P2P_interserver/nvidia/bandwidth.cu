// Copyright (c) 2024 BAAI. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License")
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <nccl.h>
#include <mpi.h>

#define SIZE (1024ULL * 1024ULL * 1024ULL * sizeof(float))
#define WARMUP_ITERATIONS 100
#define ITERATIONS 2000

void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void checkNcclError(ncclResult_t result, const char *msg) {
    if (result != ncclSuccess) {
        fprintf(stderr, "NCCL Error: %s: %s\n", msg, ncclGetErrorString(result));
        exit(EXIT_FAILURE);
    }
}

void checkMPIError(int result, const char *msg) {
    if (result != MPI_SUCCESS) {
        char error_string[MPI_MAX_ERROR_STRING];
        int length;
        MPI_Error_string(result, error_string, &length);
        fprintf(stderr, "MPI Error: %s: %s\n", msg, error_string);
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char **argv) {
    float *d_tensor;
    hipEvent_t start, end;
    float elapsed_time;

    checkMPIError(MPI_Init(&argc, &argv), "MPI_Init");
    int rank, nranks;
    checkMPIError(MPI_Comm_rank(MPI_COMM_WORLD, &rank), "MPI_Comm_rank");
    checkMPIError(MPI_Comm_size(MPI_COMM_WORLD, &nranks), "MPI_Comm_size");
    checkCudaError(hipSetDevice(0), "hipSetDevice");

    ncclComm_t comm;
    hipStream_t stream;

    ncclUniqueId id;
    if (rank == 0) {
        checkNcclError(ncclGetUniqueId(&id), "ncclGetUniqueId");
    }
    MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD);

    checkNcclError(ncclCommInitRank(&comm, nranks, id, rank), "ncclCommInitRank");
    checkCudaError(hipStreamCreate(&stream), "hipStreamCreate");
    
    checkCudaError(hipMalloc(&d_tensor, SIZE), "hipMalloc");

    checkCudaError(hipEventCreate(&start), "hipEventCreate");
    checkCudaError(hipEventCreate(&end), "hipEventCreate");

    printf("Rank %d: Running...\n", rank);
    checkNcclError(ncclGroupStart(), "ncclGroupStart");
    for (int i = 0; i < WARMUP_ITERATIONS; ++i) {
        if (rank == 0) {
            checkNcclError(ncclSend(d_tensor, SIZE / sizeof(float), ncclFloat, 1, comm, stream), "ncclSend");
        }
        else if (rank == 1){
            checkNcclError(ncclRecv(d_tensor, SIZE / sizeof(float), ncclFloat, 0, comm, stream), "ncclRecv");
        }
        printf("Rank %d: Warmup iteration %d\n", rank, i);
    }
    checkNcclError(ncclGroupEnd(), "ncclGroupEnd");
    checkCudaError(hipStreamSynchronize(stream), "hipStreamSynchronize");
    printf("Rank %d: Warmup done\n", rank);
    checkMPIError(MPI_Barrier(MPI_COMM_WORLD), "MPI_Barrier");

    printf("Rank %d: Running...\n", rank);
    checkCudaError(hipEventRecord(start), "hipEventRecord");
    checkNcclError(ncclGroupStart(), "ncclGroupStart");
    for (int i = 0; i < ITERATIONS; ++i) {
        if (rank == 0) {
            checkNcclError(ncclSend(d_tensor, SIZE / sizeof(float), ncclFloat, 1, comm, stream), "ncclSend");
        }
        else if (rank == 1){
            checkNcclError(ncclRecv(d_tensor, SIZE / sizeof(float), ncclFloat, 0, comm, stream), "ncclRecv");
        }
        printf("Rank %d: Iteration %d\n", rank, i);
    }
    checkNcclError(ncclGroupEnd(), "ncclGroupEnd");
    checkCudaError(hipStreamSynchronize(stream), "hipStreamSynchronize");
    checkMPIError(MPI_Barrier(MPI_COMM_WORLD), "MPI_Barrier");
    printf("Rank %d: Done\n", rank);
    checkCudaError(hipEventRecord(end), "hipEventRecord");
    printf("Rank %d: Recording done\n", rank); 
    checkCudaError(hipEventSynchronize(end), "hipEventSynchronize");
    printf("Rank %d: Synchronization done\n", rank);
    checkCudaError(hipEventElapsedTime(&elapsed_time, start, end), "hipEventElapsedTime");
    printf("Rank %d: Elapsed time: %.2fms\n", rank, elapsed_time);

    double bandwidth = SIZE * ITERATIONS / (elapsed_time / 1000.0);
    printf("[FlagPerf Result]interconnect-MPI_intraserver-bandwidth=%.2fGiB/s\n", bandwidth / (1024.0 * 1024.0 * 1024.0));
    printf("[FlagPerf Result]interconnect-MPI_intraserver-bandwidth=%.2fGB/s\n", bandwidth / (1000.0 * 1000.0 * 1000.0));

    checkCudaError(hipEventDestroy(start), "hipEventDestroy");
    checkCudaError(hipEventDestroy(end), "hipEventDestroy");
    checkCudaError(hipFree(d_tensor), "hipFree");
    checkNcclError(ncclCommDestroy(comm), "ncclCommDestroy");
    checkCudaError(hipStreamDestroy(stream), "hipStreamDestroy");
    checkMPIError(MPI_Finalize(), "MPI_Finalize");
    return 0;
}
