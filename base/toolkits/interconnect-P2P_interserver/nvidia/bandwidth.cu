// Copyright (c) 2024 BAAI. All rights reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License")
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <nccl.h>
#include <mpi.h>

#define SIZE (1024ULL * 1024ULL * 1024ULL * sizeof(float))
#define WARMUP_ITERATIONS 100
#define ITERATIONS 200

void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void checkNcclError(ncclResult_t result, const char *msg) {
    if (result != ncclSuccess) {
        fprintf(stderr, "NCCL Error: %s: %s\n", msg, ncclGetErrorString(result));
        exit(EXIT_FAILURE);
    }
}

void checkMPIError(int result, const char *msg) {
    if (result != MPI_SUCCESS) {
        char error_string[MPI_MAX_ERROR_STRING];
        int length;
        MPI_Error_string(result, error_string, &length);
        fprintf(stderr, "MPI Error: %s: %s\n", msg, error_string);
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char **argv) {
    float *d_tensor;
    hipEvent_t start, end;
    float elapsed_time;

    MPI_Init(&argc, &argv);
    int rank, nranks;
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &nranks);
    checkCudaError(hipSetDevice(rank), "hipSetDevice");

    ncclComm_t comm;
    hipStream_t stream;

    ncclUniqueId id;
    if (rank == 0) {
        checkNcclError(ncclGetUniqueId(&id), "ncclGetUniqueId");
    }
    MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD);

    checkNcclError(ncclCommInitRank(&comm, nranks, id, rank), "ncclCommInitRank");
    checkCudaError(hipStreamCreate(&stream), "hipStreamCreate");
    
    checkCudaError(hipMalloc(&d_tensor, SIZE), "hipMalloc");

    checkCudaError(hipEventCreate(&start), "hipEventCreate");
    checkCudaError(hipEventCreate(&end), "hipEventCreate");

    printf("Rank %d: Running...\n", rank);
    for (int i = 0; i < WARMUP_ITERATIONS; ++i) {
        if (rank == 0) {
            checkNcclError(ncclSend(d_tensor, SIZE / sizeof(float), ncclFloat, 1, comm, stream), "ncclSend");
        }
        else if (rank == 1){
            checkNcclError(ncclRecv(d_tensor, SIZE / sizeof(float), ncclFloat, 0, comm, stream), "ncclRecv");
        }
        printf("Rank %d: Warmup iteration %d\n", rank, i);
        checkCudaError(hipStreamSynchronize(stream), "hipStreamSynchronize");
    }
    printf("Rank %d: Warmup done\n", rank);

    MPI_Barrier(MPI_COMM_WORLD);

    printf("Rank %d: Running...\n", rank);
    checkCudaError(hipEventRecord(start), "hipEventRecord");
    for (int i = 0; i < ITERATIONS; ++i) {
        if (rank == 0) {
            checkNcclError(ncclSend(d_tensor, SIZE / sizeof(float), ncclFloat, 1, comm, stream), "ncclSend");
        }
        else if (rank == 1){
            checkNcclError(ncclRecv(d_tensor, SIZE / sizeof(float), ncclFloat, 0, comm, stream), "ncclRecv");
        }
        printf("Rank %d: Iteration %d\n", rank, i);
        checkCudaError(hipStreamSynchronize(stream), "hipStreamSynchronize"); 
    }
    MPI_Barrier(MPI_COMM_WORLD);
    printf("Rank %d: Done\n", rank);

    checkCudaError(hipEventRecord(end), "hipEventRecord"); 
    checkCudaError(hipEventSynchronize(end), "hipEventSynchronize");
    checkCudaError(hipEventElapsedTime(&elapsed_time, start, end), "hipEventElapsedTime");


    double bandwidth = SIZE * ITERATIONS / (elapsed_time / 1000.0);
    printf("[FlagPerf Result]interconnect-MPI_intraserver-bandwidth=%.2fGiB/s\n", bandwidth / (1024.0 * 1024.0 * 1024.0));
    printf("[FlagPerf Result]interconnect-MPI_intraserver-bandwidth=%.2fGB/s\n", bandwidth / (1000.0 * 1000.0 * 1000.0));

    checkCudaError(hipEventDestroy(start), "hipEventDestroy");
    checkCudaError(hipEventDestroy(end), "hipEventDestroy");
    checkCudaError(hipFree(d_tensor), "hipFree");
    checkNcclError(ncclCommDestroy(comm), "ncclCommDestroy");
    checkCudaError(hipStreamDestroy(stream), "hipStreamDestroy");
    MPI_Finalize();    
    return 0;
}
